#include "hip/hip_runtime.h"
/*
 * singleGpuSpectrometer
 * 
 * Version 2.0, April 12 2010
 *
 * This program was written by Hirofumi Kondo at the Supercomputing Engineering Laboratory,
 * Graduate School of Information Science and Technology, Osaka University, Japan.
 *
 * Copyright 2010 Supercomputing Engineering Laboratory, Graduate School of Information
 * Science and Technology, Osaka University, Japan
 *
 *
 * Compile : 
 *   nvcc -o singleGpuSpectrometer singleGpuSpectrometer.cu -I /usr/local/cuda/NVIDIA_GPU_Computing_SDK/common/inc
 *                                                             /usr/local/cuda/NVIDIA_GPU_Computing_SDK/C/lib/libcutil.a
 *                                                          -L /usr/local/cuda/lib -l cufft
 *
 * Usage : ./singleGpuSpectrometer [options]
 *   -length           : signal length of this spectrometer handle in M-points 
 *   -boxcar           : length of boxcar for smoothing
 *   -threshold        : value of threshold
 *   -max_detect_point : value of maximum detected points over threshold in each boxcar
 *   -output_file      : filename of output file
 *
 * Output file format :
 *   The file format is binary format.
 *   The output file records all spikes whose power exceed (boxcar_mean) * (threashold).
 *   The file contains 3 data
 *     1) index of signal
 *     2) the power of signal
 *     3) mean power of boxcar which the signal is in
 * 
 * Special Instruction
 *   1) Memory capacity
 *     The memory capacity that this GPU spectrometer requires is changed by the signal length.
 *     If you want to analyze 128M-points signal, GPU has to have 4GB VRAM.
 *     The maximum length that 1GB VRAM GPU can handle is 32M-points.
 *
 *   2) CUDA
 *     We recommend that you use CUDA 2.3 and CUFFT 2.3.
 *     This is not necessary condition.
 *     But the execution time is wrong if you use CUDA 2.2 and CUFFT 2.2.
 */


// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>

// includes, project
#include <cutil_inline.h>
#include "pasp.h"
#include "pasp.c"
#include "pasp_config.h"
#include "pasp_process.h"

/*
 * typedef
 */
typedef struct {
	int           index;
	float         power;
	float         mean;
} outputStruct;



/*
 * define constant value
 */
#define FILENAME_BUFSIZE 200
#define SUM_MAX_THREAD   256
#define SUB_MAX_THREAD   256
#define MAX_THREAD       256
#define MAX_GRID         32*1024
#define LOOP_NUM         1



/*
 * global variable
 */
// default length of singal data that this program analyze
int  signalLength = 1024 * 1024 * 16;

// default value of boxcar, this can be change by -boxcar argument
int  boxcar = 8192;

// default value of threshold, this can be changed by -threshold argument
int  threshold = 20;

// default filename of output file, this can be changed by -output_file argument
char outputFileName[FILENAME_BUFSIZE] = "report.txt";

// the memory size and pointer for generated signal data
unsigned int hostSignalDataMemSize = 0;
char         *hostSignalData = NULL;

// the memory size and pointer for output data
unsigned int hostOutputDataMemSize = 0;
outputStruct *hostOutputData = NULL;

// we limit the detected signal points in each boxcar.
// 'maximumDetectPointInBoxcar' specify its value, this can be changed by -max_detect_point argument
int maximumDetectPointInBoxcar = 16;

// output file 
int   outputCounter = 0;
int   outputFclosePeriod = 10;
FILE *outputFilePointer = NULL;


/*
 * include other source
 */
#include "kernelExec.cu"
#include "fourStepFFT.cu"
#include "output.c"


/* 
 * Prototype declaration
 */
void parse_args(int, char**);
void init_host();
void terminate_host();
void generate_signal();
void do_analyze_on_gpu();



/*
 * Program main
 */
int main(int argc, char** argv){
 
	// parse the arguments
	parse_args(argc, argv);

	// initialize host memory
	init_host();

	// generate signal
	generate_signal();

	// Analyze signal on GPU
	do_analyze_on_gpu();

	// free host memory
	terminate_host();

	// exit program
	cutilExit(argc, argv);
}



/*
 * init_host();
 */
void init_host(){
	hostSignalDataMemSize = sizeof(char) * signalLength * 2;
	hostOutputDataMemSize = sizeof(outputStruct) * maximumDetectPointInBoxcar * (signalLength / boxcar);

	cutilSafeCall( hipHostMalloc( (void**)&hostSignalData, hostSignalDataMemSize));
	if(hostSignalData==NULL){
		fprintf(stderr,"Error : hipHostMalloc failed\n");
		exit(-1);
	}

	cutilSafeCall( hipHostMalloc( (void**)&hostOutputData, hostOutputDataMemSize));
	if(hostOutputData==NULL){
		fprintf(stderr,"Error : hipHostMalloc failed\n");
		exit(-1);
	}

	return;
}



/*
 * parse_args
 */
void parse_args(int argc, char** argv) {
    for (int i=1;i<argc;i++) {
        if (!strcmp(argv[i], "-length")) {
            signalLength = atoi(argv[++i]) * 1024 * 1024;
        } else if (!strcmp(argv[i], "-boxcar")){
			boxcar = atoi(argv[++i]);
		} else if (!strcmp(argv[i], "-threshold")){
			threshold = atoi(argv[++i]);
		} else if (!strcmp(argv[i], "-max_detect_point")){
			maximumDetectPointInBoxcar = atoi(argv[++i]);
		} else if (!strcmp(argv[i], "-output_file")){
			strncpy(outputFileName, argv[++i], FILENAME_BUFSIZE);
			if(outputFileName[FILENAME_BUFSIZE-1]!='\0'){
				fprintf(stderr,"Error : Too long output file name. maximum length = %d\n", FILENAME_BUFSIZE-1);
				exit(-1);
			}
		} else {
			fprintf(stderr,"Error : wrong argument\n");
        }
    }

	return;
}



/*
 * terminate_host()
 */
void terminate_host(){
	
	// Free signal data memory
	hipHostFree(hostSignalData);
	hostSignalData = NULL;

	// Free output data memory
	hipHostFree(hostOutputData);
	hostOutputData = NULL;

	return;
}



/*
 * generate_signal()
 */
void generate_signal(){



//	srand((int) time(NULL));

//	for(int i=0; i<signalLength*2; i++){
//		hostSignalData[i] = ((rand() % 16) - 8);
//	}


/*    
	FILE *bin_file;
	bin_file=fopen("file.bin","rb");
	int buffer,bytesread;
    
	for(int i=0; i<signalLength*2; i++){

		if(bytesread != 0){
	    
		    fread(&buffer,4,1,bin_file);
		    hostSignalData[i] = ((buffer % 16) - 8);
		}
	}
	
	fclose(bin_file);
*/


    for(int i=0; i<signalLength; i++){
	hostSignalData[2*i] = (int) 127*sinf(4* i * 8 * 2*3.14159265/(float)signalLength);
	hostSignalData[2*i+1] = (int) 127*cosf(4* i * 8 * 2*3.14159265/(float)signalLength);
    }



//=== Get FIFO Dada ===//


printer();
/*

    int fifo;
    int ret;
    struct sigaction newact;

    //create the fifo
    debug_fprintf(stderr, "Creating fifo %s\n", RAW_UDP_FILE_NAME);
    ret = mkfifo(RAW_UDP_FILE_NAME,0666);
    if(ret == -1)
    {
        if(errno == EEXIST)
        {
            debug_fprintf(stderr, "File already exists. Will attempt to open.\n");
        }
        else
        {
            perror("Error creating fifo");
            exit(1);
        }
    }

    //open the fifo
    debug_fprintf(stderr, "Opening fifo %s\n", RAW_UDP_FILE_NAME);
    fifo = open(RAW_UDP_FILE_NAME, O_WRONLY);
    if(fifo == -1)
    {
        perror("Error opening fifo");
        exit(1);
    }

    //set up the signal handler
    newact.sa_handler = cleanup;
    sigemptyset(&newact.sa_mask);
    newact.sa_flags = 0;

    //start listening for Ctrl-C
    sigaction(SIGINT, &newact, NULL);

    //receive packets and write into the fifos
    receive_packets(fifo);

    debug_fprintf(stderr, "Closing fifo\n");


    close(fifo);
*/
/////////////////////////
}


/*
 * do_analyze_on_gpu
 */
void do_analyze_on_gpu(){
	
	// Device memory pointer
	char         *devSignalData = NULL;
	hipfftComplex *devFFTData    = NULL;
	float        *devPowerData  = NULL;
	float        *devAvgRe      = NULL;
	float        *devAvgIm      = NULL;
	float        *devPartSumRe  = NULL;
	float        *devPartSumIm  = NULL;
	outputStruct *devOutputData = NULL;

	// Memory size for device
	unsigned int devSignalDataMemSize = 0;
	unsigned int devFFTDataMemSize    = 0;
	unsigned int devPowerDataMemSize  = 0;
	unsigned int devPartSumMemSize    = 0;
	unsigned int devOutputDataMemSize = 0;


	// Calculate memory size
	devSignalDataMemSize = hostSignalDataMemSize;
	devFFTDataMemSize    = sizeof(hipfftComplex) * signalLength;
	devPowerDataMemSize  = sizeof(float) * signalLength;
	devPartSumMemSize    = sizeof(float) * SUM_MAX_THREAD;
	devOutputDataMemSize    = hostOutputDataMemSize;

	// Allocate device memory
	cutilSafeCall( hipMalloc( (void**) &devSignalData, devSignalDataMemSize) );
	cutilSafeCall( hipMalloc( (void**) &devFFTData,    devFFTDataMemSize) );
	cutilSafeCall( hipMalloc( (void**) &devPowerData,  devPowerDataMemSize) );
	cutilSafeCall( hipMalloc( (void**) &devPartSumRe,  devPartSumMemSize) );
	cutilSafeCall( hipMalloc( (void**) &devPartSumIm,  devPartSumMemSize) );
	cutilSafeCall( hipMalloc( (void**) &devAvgRe, sizeof(float) * 1) );
	cutilSafeCall( hipMalloc( (void**) &devAvgIm, sizeof(float) * 1) );
	cutilSafeCall( hipMalloc( (void**) &devOutputData, devOutputDataMemSize) );

	// the row length and col length of matrix
	int matrixX, matrixY;

	// the value of 'matrixY' must be fixed!!! because this program includes only 16-point fft kernel.
	matrixY = 16;
	matrixX = signalLength / matrixY;

	// Initialize output file
	init_output_file();

	// timer
	unsigned int timer;
	cutCreateTimer(&timer);	


	// Main loop
	for(int iter=0; iter<LOOP_NUM; iter++){

		// timer
		cutResetTimer(timer);
		cutStartTimer(timer);

		// CPU -> GPU : move signal data from host to device
		cutilSafeCall( hipMemcpy(devSignalData, hostSignalData, devSignalDataMemSize, hipMemcpyHostToDevice));
		hipDeviceSynchronize();

		// GPU : convert char format signal data to float format
		convert_to_float_exec(devSignalData, devPartSumRe, devPartSumIm, devAvgRe, devAvgIm, devFFTData);

		// GPU : do fft
		do_four_step_fft(devFFTData, devPowerData, matrixX, matrixY);

		// GPU : detect strong power spectrum
		calc_over_threshold_exec(devPowerData, devOutputData);

		// GPU -> CPU : copy detect spectrum data from device to host
		cutilSafeCall( hipMemcpy( hostOutputData, devOutputData, devOutputDataMemSize, hipMemcpyDeviceToHost));
		hipDeviceSynchronize();

		// CPU : output detect power spectrum to file
		output_spectrum(hostOutputData, iter, 1);

		// timer
		cutStopTimer(timer);
		printf("time = %f, %d done...\n",cutGetTimerValue(timer), iter);

	}

	// Terminate output file
	terminate_output_file();


	// Free device memory
	cutilSafeCall( hipFree( devSignalData ) );
	cutilSafeCall( hipFree( devFFTData ) );
	cutilSafeCall( hipFree( devPowerData ) );
	cutilSafeCall( hipFree( devPartSumRe ) );
	cutilSafeCall( hipFree( devPartSumIm ) );
	cutilSafeCall( hipFree( devAvgRe ) );
	cutilSafeCall( hipFree( devAvgIm ) );
}
